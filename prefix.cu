#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define NUM_BLOCKS 1
#define BLOCK_SIZE 256
#define NUM_MEM 32768

//stage one
__global__ void prefixOne(int *in_array, int *out_array, int unsize, int size)
{	
	int tid = threadIdx.x;
	for(int j = 0; j < unsize; j++){
		if (j == 0){
			out_array[tid * unsize] = in_array[tid * unsize];
		} else {
			for(int k = 0; k <= j; k++) {
				out_array[tid * unsize + j] += in_array[tid * unsize + k];
			}
		}
	}
}

//stage two
__global__ void prefixTwo(int *in_array, int unsize, int maxid, int idx)
{
	int tid = threadIdx.x;
	if (tid <= maxid) {
		int maxstep = unsize * (int)(powf(2, idx - 1));
		for(int j = 0; j < maxstep; j++) {
			int startIdx = unsize * (int)(powf(2, idx - 1)) * (1 + 2 * tid);
			in_array[startIdx + j] = in_array[startIdx - 1] + in_array[startIdx + j];
		}		
	}
	
	
}

void prefixsum(int blocks, int threads, int steps, int *array_h, int size)
{
	int *array_d;
	int *tmp_one;
	int unsize = size/(blocks * threads);
	
	
	dim3 dim_grid(blocks, 1, 1);
	dim3 dim_block(threads, 1, 1);

	// allocate tmp_d
	hipMalloc((void **)&tmp_one, size * sizeof(int));
	//cudaMalloc((void **)&out_array_d, blocks * sizeof(int));
	hipMalloc((void **)&array_d, size * sizeof(int));
	//copy data from host to device
	hipMemcpy(array_d, array_h, size * sizeof(int),
		   hipMemcpyHostToDevice);
	
	hipMemset(tmp_one, 0, size * sizeof(int));
	//do stage 1
	prefixOne<<<dim_grid, dim_block>>> (array_d, tmp_one, unsize, size);
	
	if (steps !=0) {
		int maxtid = 0;
		//do stage 2
		for (int i = 1; i <= steps; i++) {
			maxtid = (int)pow(2, steps-i) - 1;
			prefixTwo<<<dim_grid, dim_block>>>(tmp_one, unsize, maxtid, i);
		}
	}
	
		hipMemcpy(array_h, tmp_one, size * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(array_d);
	hipFree(tmp_one);
}



void prepare_numbers(int **array, int count)
{
	int *numbers = (int *)malloc(count * sizeof(int));

	// load array
	for (int i = 0; i < count; i++) {
		numbers[i] = 1;
	}

	*array = numbers;
}

void print_array(int *array, int count)
{
	for (int i = 0; i < count; i++) {
		printf("%d\t", array[i]);
	}
	printf("\n");
}

int main()
{
	int blocks, threads, max, stepTwo;
	int *array;
    float calTime;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	blocks = NUM_BLOCKS;
	threads = BLOCK_SIZE;
	stepTwo = 8;
	max = NUM_MEM;

	// pre-init numbers
	array = NULL;
	prepare_numbers(&array, max);

	hipEventRecord(start, 0);
	prefixsum(blocks, threads, stepTwo, array, max);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&calTime, start, end);

	// print array
//	print_array(array, max);
	printf("the elapsed time with %d threads is %.10f\n", threads, calTime);

	free(array);

	return 0;
}
